#include "hip/hip_runtime.h"
#define WIN32_LEAN_AND_MEAN      // Exclude rarely-used stuff from Windows headers

#include <windows.h>
#include "hip/hip_runtime.h"
//#include ""
#include "hipfft/hipfft.h"
#include <fstream>
#include <filesystem>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>
#include <sysinfoapi.h>
#include <iostream>
#include <sstream>
#include <iostream>
#include <windows.h>
#include <stdlib.h>
#include <algorithm>
#include <iomanip>
#include<direct.h>
#include <map>
//#include <limits>

using namespace std;



// Complex data type
typedef hipfftDoubleComplex Complex;//hipfftComplex
typedef std::map<pair<int, int>, int> map_t;


///////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////// USER SETTINGS //////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////

// FFT data
string g_program_version = "1.0.0.8";

const unsigned int meshSizeX = 1024;//4;//32 196;
const unsigned int meshSizeY = 1024;//4;//32 196;

string g_U_matrix_fileName = "U10.txt";
string g_E_matrix_fileName = "E10_0.txt";
string g_D_matrix_fileName = "D10_10000.txt";
string g_W_matrix_fileName = "W8_1cycle.txt";
string g_WD_matrix_fileName = "WD8_1cycle.txt";
string g_G_fileName = "G10.txt";
string g_Indxs_fileName = "indxs10.txt";
string g_UL_filename = "UL10_10000_E0.txt";
int g_PrintPropogateMatrixStepNum = 0; //Step number to save Propogate matrix. If value == -1 then not used at all.
string g_P_matrix_at_step = "P10_first_step.txt"; //filename to save P matrix at step defined by g_PrintPropogateMatrixStepNum
string g_UL_matrix_at_step = "UL10_first_step.txt";//filename to save UL matrix at step defined by g_PrintPropogateMatrixStepNum



int g_number_of_cycles = 10000;

bool g_LogMultiplicationHost = false;


///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////

const unsigned int meshSize1D = meshSizeX * meshSizeY;
const int g_MatrixMemSize = meshSizeX * meshSizeY * sizeof(Complex);
const unsigned int ThreadsInBlock = 1024;// meshSizeX;

Complex *d_h0 = 0;
Complex *d_U = 0;
Complex *d_W = 0;
Complex *d_WD = 0;
Complex *d_Difract = 0;
Complex *d_Uinverse = 0;
Complex *d_E = 0;
Complex *d_Temp = 0;
Complex* d_Temp2 = 0;
Complex *d_P = 0;
//Complex *g_h_difract = NULL;
bool g_UseMultiplication = true;
hipfftHandle fftPlan;




Complex h_U[meshSizeX][meshSizeY];
Complex h_U_original[meshSizeX][meshSizeY];
Complex h_Uout[meshSizeX][meshSizeY];
Complex h_Difract[meshSizeX][meshSizeY];
Complex h_E[meshSizeX][meshSizeY];
Complex h_P[meshSizeX][meshSizeY];
Complex h_W[meshSizeX][meshSizeY];
Complex h_G[meshSizeX][meshSizeY];
std::map<pair < int, int >, int> h_Indxs;


//Executes Forward transform on p_inMatrix and returns result in p_outMatrix
void ExecC2CForward(Complex (&p_inMatrix)[meshSizeX][meshSizeY], Complex (&p_outMatrix)[meshSizeX][meshSizeY]);
void ExecC2CForward(Complex** p_inMatrix, Complex** p_outMatrix);

//Executes Inverse transform on p_inMatrix and returns result in p_outMatrix
void ExecC2CInverse(Complex (&p_inMatrix)[meshSizeX][meshSizeY], Complex (&p_outMatrix)[meshSizeX][meshSizeY]);

//Executes Forward transform on p_inMatrix1, multiplies result of p_inMatrix1 and global difract matrix, returns Inversed result in p_outMatrix
void ExecC2CForwardInverse(Complex(&p_inMatrix)[meshSizeX][meshSizeY], Complex(&p_outMatrix)[meshSizeX][meshSizeY]);

void ExecC2CForward1D(Complex(&p_inMatrix)[meshSize1D], Complex(&p_outMatrix)[meshSize1D]);

//Compare matrixes
bool CompareMatrix(Complex(&p_MatrixA)[meshSizeX][meshSizeY], Complex(&p_MatrixB)[meshSizeX][meshSizeY], string &pCompareError);



void CheckError(bool p_Yes, string p_Message);
void printfComplexVector(const Complex *pVec, int pSize);
void printfFloatVector(const float *pVec, int pSize);
void print2DArray(Complex(&p_inMatrix)[meshSizeX][meshSizeY], int p_Xsize, int p_Ysize, int p_Scale = 1);
void print2DArray(Complex* (p_inMatrix[meshSizeX][meshSizeY]), int p_Xsize, int p_Ysize, int p_Scale = 1);
void print2DArray(Complex** p_inMatrix, int p_Xsize, int p_Ysize, int p_Scale = 1);

void Transform2Dto1D(Complex(&p_inMatrix)[meshSizeX][meshSizeY], Complex(&p_outMatrix)[meshSizeX * meshSizeY]);
void Transform1Dto2D(Complex(&p_inMatrix)[meshSizeX * meshSizeY], Complex(&p_outMatrix)[meshSizeX][meshSizeY]);
void read2DArrayFromFile(Complex(&p_inMatrix)[meshSizeX][meshSizeY], string p_FileName);
void read2DArrayFromFile(Complex **p_inMatrix, string p_FileName);
void readIndxMapFromFile(std::map<pair < int, int >, int> &pMap, string p_FileName);
void write2DArrayToFile(Complex(&p_inMatrix)[meshSizeX][meshSizeY], string p_FileName, int p_Scale = 1);
void write2DArrayToFile(Complex** p_inMatrix, int p_meshSizeX, int p_meshSizeY, string p_FileName);

hipError_t mulComplexWithCuda(float2* c, const float2* a, const float2* b, unsigned int size);
void print1DArray(Complex(&p_inMatrix)[meshSize1D], int p_PrintSizeBegin, int p_PrintSizeEnd);
void MemSet1DArray(Complex(&p_inMatrix)[meshSize1D], float p_XVal, float p_YVal);
void MemSet1DArray(Complex* p_inMatrix, int p_Size, float p_XVal, float p_YVal);
void MemSet2DArray(Complex (&p_inMatrix)[meshSizeX][meshSizeY], float p_XVal, float p_YVal);
void RefreshPropogateMatrix(int p_StepCount, Complex(&p_inMatrix)[meshSizeX][meshSizeY]);
void NormalyzeFFT(Complex(&p_inMatrix)[meshSizeX][meshSizeY], int p_Scale);
string getCurrentDirectoryOnWindows();



int RunStack2DTest();
int RunHeap1DTest();
void Initialization();
void Finalization();

void CreateGenericMarix();
const Complex ComplexDiv(const Complex& x, const Complex& y);

void HostMatrixMultiplication(Complex(&p_Result)[meshSizeX][meshSizeY], Complex(&p_W)[meshSizeX][meshSizeY],
    Complex(&p_Difract)[meshSizeX][meshSizeY]);

void LogMatrixMultiplication(Complex(&p_Host)[meshSizeX][meshSizeY], Complex(&p_Device)[meshSizeX][meshSizeY]);



inline void CheckError(bool p_Yes, string p_Message)
{
    if (!p_Yes)
        throw std::exception(p_Message.c_str());
}

static __global__ void mulKernelComplex(int n, Complex *c, const Complex *a, const Complex *b, bool p_UseMultiplication)
{
    //int i = threadIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
    {
        if (p_UseMultiplication)
        {
            c[i].x = (a[i].x * b[i].x) - (a[i].y * b[i].y);
            c[i].y = (a[i].x * b[i].y) + (a[i].y * b[i].x);

        }
        else
        {
            c[i].x = a[i].x;
            c[i].y = a[i].y;
        }
    }
}

static __global__ void mulKernelPropogate(int n, Complex* c, const Complex* a, const Complex* b)
{
    //int i = threadIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
    {
        c[i].x = b[i].x;//(a[i].x * b[i].x) - (a[i].y * b[i].y);
        c[i].y = b[i].y;//(a[i].x * b[i].y) + (a[i].y * b[i].x);
    }
}

static __global__ void divKernelComplex(int n, Complex* c, const Complex* a, int pVal)
{
    //int i = threadIdx.x;
    //int j = threadIdx.y;
    /* for (int i = 0; i < n; i++)
    {
        c[i].x = a[i].x / pVal;
        c[i].y = a[i].y / pVal;
    }
    */

    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
    {
        c[i].x = a[i].x / pVal;// 999;
        c[i].y = a[i].y / pVal; //999;
    }
}



hipError_t cudaStatus;


int main()
{
    try
    {
        printf("program version: %s\n", g_program_version.c_str());
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            return -1;
        }

        printf("matrix %d x %d\n", meshSizeX, meshSizeY);
        printf("size of Complex %d\n", sizeof(Complex));
        printf("memory to allocate for device variable matrix %d \n\n", g_MatrixMemSize);
        printf("number of circles %d\n", g_number_of_cycles);

        printf("U matrix fileName = %s\n", g_U_matrix_fileName.c_str());
        printf("D matrix fileName = %s\n", g_D_matrix_fileName.c_str());

        printf("E matrix fileName = %s\n", g_E_matrix_fileName.c_str());
        printf("G matrix fileName = %s\n", g_G_fileName.c_str());
        printf("Indxs fileName = %s\n", g_Indxs_fileName.c_str());


        if (g_number_of_cycles == 1)
        {
            printf("W matrix output fileName = %s\n", g_W_matrix_fileName.c_str());
            printf("WD matrix output fileName = %s\n", g_WD_matrix_fileName.c_str());
        }


        printf("UL matrix output fileName = %s\n", g_UL_filename.c_str());



        // Creating a directory

        string a_OutputDir = getCurrentDirectoryOnWindows() + "//Output";
        struct stat sb;
                                                      //Folder          
        if (stat(a_OutputDir.c_str(), &sb) == 0 && (sb.st_mode == 16895))
            cout << "/Output folder exists." << endl;
        else
        {
            int a_stat = mkdir(a_OutputDir.c_str());
            if (!a_stat)
                cout << "Folder /Output created " << endl;
            else
                cout << "Impossible create folder /Output" << endl;
        }


        int a_Result = 0;
        int a_UserChooice = 1;
        //   printf("please enter your choice:\n");
        //   printf("1. Use 2D array in stack\n");
        //   printf("2. Use 2D array in heap\n");
        //   printf("3. Exit\n");

       //    cin >> a_UserChooice;
        printf("Use multiplication? y/n:\n");
        string a_Use = "n";
        cin >> a_Use;

        transform(a_Use.begin(), a_Use.end(), a_Use.begin(), ::tolower);
        g_UseMultiplication = a_Use == "y";

        if (a_UserChooice == 1)
            a_Result = RunStack2DTest();
        if (a_UserChooice == 2)
            a_Result = RunHeap1DTest();
        else
            a_Result = 0;

        Finalization();
        /*printf("Create Generic Matrix y/n?:\n");
        cin >> a_Use;
        if( a_Use == "y" )
            CreateGenericMarix();
    */
        return a_Result;
    }
    catch (std::exception& e) 
    {
       std::cerr << e.what() << std::endl;
    }
};

int RunStack2DTest()
{

    Initialization();

    MemSet2DArray(h_P, 0, 0);
    MemSet2DArray(h_U, 0, 0);
    MemSet2DArray(h_Uout, 0, 0);
    MemSet2DArray(h_Difract, 0, 0);
    MemSet2DArray(h_E, 0, 0);
    MemSet2DArray(h_G, 0, 0);


    printf("reading G matrix %s\n", g_G_fileName.c_str());
    read2DArrayFromFile(h_G, g_G_fileName);

    printf("reading Indxs map %s\n", g_G_fileName.c_str());
    readIndxMapFromFile(h_Indxs, g_Indxs_fileName);

    printf("input matrix %s\n", g_U_matrix_fileName.c_str());
    read2DArrayFromFile(h_U, g_U_matrix_fileName);
    print2DArray(h_U, 4, 4);

    if (g_UseMultiplication)
    {
        checkCudaErrors(hipMalloc((void**)&d_Difract, g_MatrixMemSize));
        printf("reading difract. %s\n", g_D_matrix_fileName.c_str());
        read2DArrayFromFile(h_Difract, g_D_matrix_fileName);
        checkCudaErrors(hipMemcpy(d_Difract, h_Difract, g_MatrixMemSize, hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc((void**)&d_E, g_MatrixMemSize));
        printf("reading E matrix. %s\n", g_E_matrix_fileName.c_str());
        read2DArrayFromFile(h_E, g_E_matrix_fileName);
        checkCudaErrors(hipMemcpy(d_E, h_E, g_MatrixMemSize, hipMemcpyHostToDevice));
    }


    /// begin

    checkCudaErrors(hipMemcpy(d_U, h_U, g_MatrixMemSize, hipMemcpyHostToDevice));


    div_t div_res;

    bool a_UseHostNormalization = false;
    //   printf("Input choice. \n");
    //   printf("Use Host Normalization : 1\n");
    //   printf("Use Device Normalization : 2\n");
    //   int a_Choice = 0;
   //    cin >> a_Choice;
   //    a_UseHostNormalization = (a_Choice == 1);

    bool a_ProduceIntermediateFiles = false;

    if ((g_number_of_cycles == 1) && g_UseMultiplication)
        a_ProduceIntermediateFiles = true;
///////////////////////////////////////////////////////////////////////////////
////////////////////////////// MAIN LOOP //////////////////////////////////////
    printf("calculating...%d cycles\n", g_number_of_cycles);
    DWORD a_Start = GetTickCount();
    int a_StepCount = 0;
    bool a_WriteUL_at_Step = false;
    for (int i = 0; i < g_number_of_cycles; i++)
    {

        checkCudaErrors(hipfftExecZ2Z(fftPlan, d_U, d_W, HIPFFT_FORWARD));
        if (a_ProduceIntermediateFiles)
        {
            checkCudaErrors(hipMemcpy(h_Uout, d_W, g_MatrixMemSize, hipMemcpyDeviceToHost));
            printf("output W matrix after ExecZ2ZForward  writing to %s\n", g_W_matrix_fileName.c_str());
            print2DArray(h_Uout, 4, 4, 1);
            write2DArrayToFile(h_Uout, "Output//" + g_W_matrix_fileName);
        }

        if (g_LogMultiplicationHost)
        {
            checkCudaErrors(hipMemcpy(h_Uout, d_W, g_MatrixMemSize, hipMemcpyDeviceToHost));
            HostMatrixMultiplication(h_W, h_Uout, h_Difract);
        }

        // Launch a kernel on the GPU with one thread for each element. 
                              //4
        mulKernelComplex << < 4, ThreadsInBlock >> > (meshSize1D, d_WD, d_W, d_Difract, g_UseMultiplication); //32

        if (g_LogMultiplicationHost)
        {
            checkCudaErrors(hipMemcpy(h_Uout, d_WD, g_MatrixMemSize, hipMemcpyDeviceToHost));
            LogMatrixMultiplication(h_W, h_Uout);
        }

        if (a_ProduceIntermediateFiles)
        {
            checkCudaErrors(hipMemcpy(h_Uout, d_WD, g_MatrixMemSize, hipMemcpyDeviceToHost));
            printf("output matrix ExecZ2zForward with Multiplication writing to %s\n", g_WD_matrix_fileName.c_str());
            print2DArray(h_Uout, 4, 4, 1);
            write2DArrayToFile(h_Uout, "Output//" + g_WD_matrix_fileName);
        }

        checkCudaErrors(hipfftExecZ2Z(fftPlan, d_WD, d_Uinverse, HIPFFT_BACKWARD));

        if (!a_UseHostNormalization)
        {                          //4                              // result of division    
            divKernelComplex << < 4, ThreadsInBlock >> > (meshSize1D, d_Temp, d_Uinverse, meshSize1D);//final result is in d_U 
        }
        else
        {
            checkCudaErrors(hipMemcpy(h_U, d_Uinverse, g_MatrixMemSize, hipMemcpyDeviceToHost));
            NormalyzeFFT(h_U, meshSize1D);
            checkCudaErrors(hipMemcpy(d_Temp, h_U, g_MatrixMemSize, hipMemcpyHostToDevice));//final result is in d_U
        }


        div_res = div(i, 100);
        if (div_res.rem == 0)
        {
            printf(".");
            RefreshPropogateMatrix(a_StepCount, h_P);

            if (a_StepCount == g_PrintPropogateMatrixStepNum)
            {
                printf("writing Propogate matrix at Step %d to %s\n", a_StepCount, g_P_matrix_at_step.c_str());
                write2DArrayToFile(h_P, "Output//" + g_P_matrix_at_step);
                a_WriteUL_at_Step = true;
            }
            a_StepCount++;
            checkCudaErrors(hipMemcpy(d_P, h_P, g_MatrixMemSize, hipMemcpyHostToDevice));//final result is in d_U
        }

        //mul by E0
        mulKernelComplex << < 4, ThreadsInBlock >> > (meshSize1D, d_Temp2, d_Temp, d_E, g_UseMultiplication);
        mulKernelComplex << < 4, ThreadsInBlock >> > (meshSize1D, d_U, d_Temp2, d_P, g_UseMultiplication);

        
        if (a_WriteUL_at_Step)
        {
            a_WriteUL_at_Step = false;
            checkCudaErrors(hipMemcpy(h_Uout, d_U, g_MatrixMemSize, hipMemcpyDeviceToHost));
            printf("writing UL matrix at step %d to %s\n", a_StepCount, g_UL_matrix_at_step.c_str());
            write2DArrayToFile(h_Uout, "Output//" + g_UL_matrix_at_step);
        }
        
    }

    if (!a_UseHostNormalization)//use Device normalization
        checkCudaErrors(hipMemcpy(h_U, d_U, g_MatrixMemSize, hipMemcpyDeviceToHost));



    printf("ExecZ2ZForwardInverse Timing %d ms for %d circles\n", GetTickCount() - a_Start, g_number_of_cycles);
    printf("output array ExecZ2ZForward  Mul ExecZ2Zinverse\n");
    print2DArray(h_U, 4, 4, 1);//

    if (!g_UseMultiplication)
    {
        printf("...comparing Original %s with output matrix h_U\n", g_U_matrix_fileName.c_str());
        MemSet2DArray(h_U_original, 0, 0);
        read2DArrayFromFile(h_U_original, g_U_matrix_fileName);
        string a_CompareError;
        if (!CompareMatrix(h_U_original, h_U, a_CompareError))
            printf("matrixes h_U_original and h_U not equal: %s\n\n", a_CompareError.c_str());
    }

    if (g_UseMultiplication)
    {
        printf("write output %s ?  y/n\n", g_UL_filename.c_str());
        string a_Use = "n";
        cin >> a_Use;

        if (a_Use == "y")
        {
            printf("writing %s\n", g_UL_filename.c_str());
            write2DArrayToFile(h_U, "Output//" + g_UL_filename);//meshSize1D
        }
    }
    return 0;
}

void RefreshPropogateMatrix(int p_StepCount, Complex(&p_inMatrix)[meshSizeX][meshSizeY])
{    
    int G_I, G_J;
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY;j++)
        {

            //std::map<pair<int, int>, int >
            map_t::iterator a_result = h_Indxs.find(make_pair(p_StepCount, i));
            CheckError(a_result != h_Indxs.end(), "index not found in Indxs");
            G_I = a_result->second;

            a_result = h_Indxs.find(make_pair(p_StepCount, j));
            CheckError(a_result != h_Indxs.end(), "index not found in Indxs");
            G_J = a_result->second;

            p_inMatrix[i][j].x = h_G[G_I][G_J].x;
            p_inMatrix[i][j].y = h_G[G_I][G_J].y;
            
        }
    }
    
}

void Initialization()
{
    printf("begin\n");

    checkCudaErrors(hipMalloc((void**)&d_W, g_MatrixMemSize));
    checkCudaErrors(hipMalloc((void**)&d_WD, g_MatrixMemSize));
    //propogate matrix
    checkCudaErrors(hipMalloc((void**)&d_P, g_MatrixMemSize));
    checkCudaErrors(hipMalloc((void**)&d_U, g_MatrixMemSize));
    checkCudaErrors(hipMalloc((void**)&d_Uinverse, g_MatrixMemSize));
    checkCudaErrors(hipMalloc((void**)&d_Temp, g_MatrixMemSize));
    checkCudaErrors(hipMalloc((void**)&d_Temp2, g_MatrixMemSize));
    // size_t pitch;
     // checkCudaErrors(hipMallocPitch((void**)&d_U, &pitch, meshSizeX * sizeof(Complex), meshSizeY * sizeof(Complex)));




    // create FFT plan
    checkCudaErrors(hipfftPlan2d(&fftPlan, meshSizeX, meshSizeY, HIPFFT_Z2Z));
}

void Finalization()
{
    checkCudaErrors(hipFree(d_U));
    checkCudaErrors(hipFree(d_Difract));
    checkCudaErrors(hipFree(d_W));
    checkCudaErrors(hipFree(d_WD));
    checkCudaErrors(hipFree(d_Uinverse));
    checkCudaErrors(hipFree(d_E));
    checkCudaErrors(hipFree(d_Temp));
    checkCudaErrors(hipFree(d_Temp2));
    checkCudaErrors(hipFree(d_P));

   // free(g_h_difract);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
    fprintf(stderr, "the end!");
}






//Executes Forward transform on p_inMatrix and returns result in p_outMatrix
void ExecC2CForward1D(Complex(&p_inMatrix)[meshSize1D], Complex(&p_outMatrix)[meshSize1D])
{
    checkCudaErrors(hipMemcpy(d_U, p_inMatrix, g_MatrixMemSize, hipMemcpyHostToDevice));


    checkCudaErrors(hipfftExecZ2Z(fftPlan, d_U, d_U, HIPFFT_FORWARD));


    checkCudaErrors(hipMemcpy(p_outMatrix, d_U, g_MatrixMemSize, hipMemcpyDeviceToHost));
}



void ExecC2CForward(Complex **p_inMatrix, Complex **p_outMatrix)
{
    checkCudaErrors(hipMemcpy2D(d_U, 1, p_inMatrix, 1, meshSizeX, meshSizeY, hipMemcpyHostToDevice));


checkCudaErrors(hipfftExecZ2Z(fftPlan, d_U, d_U, HIPFFT_FORWARD));


checkCudaErrors(hipMemcpy2D(p_outMatrix, 1, d_U, 1, meshSizeX, meshSizeY, hipMemcpyDeviceToHost));
}


//Executes Forward transform on p_inMatrix and returns result in p_outMatrix
void ExecC2CForward(Complex(&p_inMatrix)[meshSizeX][meshSizeY], Complex(&p_outMatrix)[meshSizeX][meshSizeY])
{
    checkCudaErrors(hipMemcpy(d_U, p_inMatrix, g_MatrixMemSize, hipMemcpyHostToDevice));


    checkCudaErrors(hipfftExecZ2Z(fftPlan, d_U, d_U, HIPFFT_FORWARD));


    checkCudaErrors(hipMemcpy(p_outMatrix, d_U, g_MatrixMemSize, hipMemcpyDeviceToHost));
}

//Executes Inverse transform on p_inMatrix and returns result in p_outMatrix
void ExecC2CInverse(Complex(&p_inMatrix)[meshSizeX][meshSizeY], Complex(&p_outMatrix)[meshSizeX][meshSizeY])
{
    checkCudaErrors(hipMemcpy(d_U, p_inMatrix, g_MatrixMemSize, hipMemcpyHostToDevice));
    //  printf("hipMemcpyHostToDevice\n");

    checkCudaErrors(hipfftExecZ2Z(fftPlan, d_U, d_U, HIPFFT_BACKWARD));
    //  printf("hipfftExecC2C  HIPFFT_BACKWARD\n");

    checkCudaErrors(hipMemcpy(p_outMatrix, d_U, g_MatrixMemSize, hipMemcpyDeviceToHost));
}

//Executes Forward transform on p_inMatrix1, multiplies result of p_inMatrix1 and global difract matrix, returns Inversed result in p_outMatrix
void ExecC2CForwardInverse(Complex(&p_inMatrix)[meshSizeX][meshSizeY], Complex(&p_outMatrix)[meshSizeX][meshSizeY])
{
    checkCudaErrors(hipMemcpy(d_U, p_inMatrix, g_MatrixMemSize, hipMemcpyHostToDevice));

    checkCudaErrors(hipfftExecZ2Z(fftPlan, d_U, d_U, HIPFFT_FORWARD));

    // Launch a kernel on the GPU with one thread for each element. 
    mulKernelComplex << <128, meshSize1D >> > (meshSize1D, d_W, d_U, d_Difract, g_UseMultiplication); //32

    ///////////////////////////////////////////////
        //For multiplication, transforming 2D matrix to 1D and back to 2D, in order to prove that mltiplication of 1D or 2D is the same reslt.
    /*
        Complex a_temp1D[meshSize * meshSize];
        Complex a_temp2D[meshSize][meshSize];

        checkCudaErrors(hipMemcpy(a_temp2D, d_ht, g_MatrixMemSize, hipMemcpyDeviceToHost));
        Transform2Dto1D(a_temp2D, a_temp1D);
        checkCudaErrors(hipMemcpy(d_ht, a_temp1D, g_MatrixMemSize, hipMemcpyHostToDevice));

        // Launch a kernel on the GPU with one thread for each element.
        mulKernelComplex <<<32, meshSize1D >>> (d_hMulResult, d_ht, d_Difract); //32

        checkCudaErrors(hipMemcpy(a_temp1D, d_hMulResult, g_MatrixMemSize, hipMemcpyDeviceToHost));
        Transform1Dto2D(a_temp1D, a_temp2D);
        checkCudaErrors(hipMemcpy(d_hMulResult, a_temp2D, g_MatrixMemSize, hipMemcpyHostToDevice));
    */
    //////////////////////////////////////////


        //checkCudaErrors(hipfftExecC2C(fftPlan, d_hMulResult, d_ht, HIPFFT_BACKWARD));



    checkCudaErrors(hipMemcpy(p_outMatrix, d_W, g_MatrixMemSize, hipMemcpyDeviceToHost));
}


void Transform2Dto1D(Complex(&p_inMatrix)[meshSizeX][meshSizeY], Complex(&p_outMatrix)[meshSizeX * meshSizeY])
{
    int a_Ind = 0;
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY;j++)
        {
            p_outMatrix[a_Ind].x = p_inMatrix[i][j].x;
            p_outMatrix[a_Ind].y = p_inMatrix[i][j].y;
            a_Ind++;
        }
    }
}


void Transform1Dto2D(Complex(&p_inMatrix)[meshSizeX * meshSizeY], Complex(&p_outMatrix)[meshSizeX][meshSizeY])
{
    int a_Ind = 0;
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY;j++)
        {
            p_outMatrix[i][j].x = p_inMatrix[a_Ind].x;
            p_outMatrix[i][j].y = p_inMatrix[a_Ind].y;
            a_Ind++;
        }
    }
}

void MemSet2DArray(Complex(&p_inMatrix)[meshSizeX][meshSizeY], float p_XVal, float p_YVal)
{
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY;j++)
        {
            p_inMatrix[i][j].x = p_XVal;
            p_inMatrix[i][j].y = p_YVal;
        }
    }
}

void MemSet1DArray(Complex(&p_inMatrix)[meshSize1D], float p_XVal, float p_YVal)
{
    for (int i = 0; i < meshSize1D; i++)
    {
        p_inMatrix[i].x = p_XVal;
        p_inMatrix[i].y = p_YVal;
    }
}

void MemSet1DArray(Complex *p_inMatrix, int p_Size, float p_XVal, float p_YVal)
{
    for (int i = 0; i < p_Size; i++)
    {
        //reinterpret_cast<Complex (*)[]>(p_inMatrix)[i].x = p_XVal;
        //reinterpret_cast<Complex (*)[]>(p_inMatrix)[i].y = p_YVal;
    }
}

void NormalyzeFFT(Complex(&p_inMatrix)[meshSizeX][meshSizeY], int p_Scale)
{
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY;j++)
        {
           // if(p_inMatrix[i][j].x > 0)
               p_inMatrix[i][j].x = p_inMatrix[i][j].x / p_Scale;
           // if(p_inMatrix[i][j].y > 0)
              p_inMatrix[i][j].y = p_inMatrix[i][j].y / p_Scale;
        }
    }

}


void print1DArray(Complex(&p_inMatrix)[meshSize1D], int p_PrintSizeBegin, int p_PrintSizeEnd)
{
    for (int i = p_PrintSizeBegin; i < p_PrintSizeEnd; i++)
    {
        printf("%f %f %d %d\n", p_inMatrix[i].x, p_inMatrix[i].y, i);
    }
}

void print2DArray(Complex (&p_inMatrix)[meshSizeX][meshSizeY], int p_Xsize, int p_Ysize, int p_Scale)
{
    for (int i = 0; i < p_Xsize; i++)
    {
        for (int j = 0; j < p_Ysize;j++)
        {            
            Complex aComp = p_inMatrix[i][j];
            printf("%f %f %d %d", aComp.x/p_Scale, aComp.y/p_Scale, i, j);
            printf("\n");
        }
    }
}

void print2DArray(Complex  **p_inMatrix, int p_Xsize, int p_Ysize, int p_Scale)
{
    for (int i = 0; i < p_Xsize; i++)
    {
        for (int j = 0; j < p_Ysize; j++)
        {
            Complex aComp = p_inMatrix[i][j];
            //Complex** a_inMatrix[meshSizeX][meshSizeY] = 
            //    reinterpret_cast<Complex **([meshSizeX][meshSizeY])>(p_inMatrix);
            printf("%f %f %d %d", aComp.x / p_Scale, aComp.y / p_Scale, i, j);
            printf("\n");
        }
    }
}

void print2DArray(Complex *(p_inMatrix[meshSizeX][meshSizeY]), int p_Xsize, int p_Ysize, int p_Scale)
{
    for (int i = 0; i < p_Xsize; i++)
    {
        for (int j = 0; j < p_Ysize;j++)
        {            
            Complex *aComp = reinterpret_cast<Complex*>(p_inMatrix[i][j]);
            printf("%f %f %d %d", aComp->x / p_Scale, aComp->y / p_Scale, i, j);
            printf("\n");
        }
    }
}



void printfComplexVector(const Complex *pVec, int pSize)
{
	for(int i = 0; i < pSize; i++)
	{
       printf("%f  %f\n", pVec[i].x, pVec[i].y);
	}
}

void printfFloatVector(const float *pVec, int pSize)
{
	for(int i = 0; i < pSize; i++)
	{
       printf("%f\n", pVec[i]);
	}
}

std::string ReplaceAll(std::string str, const std::string& from, const std::string& to) 
{
    size_t start_pos = 0;
    while ((start_pos = str.find(from, start_pos)) != std::string::npos) {
        str.replace(start_pos, from.length(), to);
        start_pos += to.length(); // Handles case where 'to' is a substring of 'from'
    }
    return str;
}

std::string GetCurrentExeDirectory()
{
    char buffer[MAX_PATH];
    GetModuleFileNameA(NULL, buffer, MAX_PATH);
    std::string::size_type pos = std::string(buffer).find_last_of("\\/");

    return std::string(buffer).substr(0, pos);
}

string getCurrentDirectoryOnWindows()
{
    string a_Result = GetCurrentExeDirectory();
    string a_BackSlash = "\\";
    string a_MySlash = "//";
    a_Result = ReplaceAll(a_Result, a_BackSlash, a_MySlash);
    return a_Result;
}

void read2DArrayFromFile(Complex(&p_inMatrix)[meshSizeX][meshSizeY], string p_FileName)
{
    string line, a_Temp;
    stringstream ss;    
    string a_Fn = getCurrentDirectoryOnWindows() + "//" + p_FileName.c_str();
    ifstream inFile(a_Fn);
    CheckError(!inFile.fail(), "read file failed :" + p_FileName);
    while (!inFile.eof())//(getline(inFile, line))
    {
        //ss << line << "\r\n";
        Complex a_Comp;
        int I, J;
        //ss >> a_Temp;
        inFile >> a_Comp.x;//  std::stof(a_Temp);
        //ss >> a_Temp;
        inFile >> a_Comp.y;// = std::stof(a_Temp);
        //ss >> a_Temp;
        inFile >> I;// = std::stoi(a_Temp);
        //ss >> a_Temp;
        inFile >> J;// = std::stoi(a_Temp);
        if ((I < meshSizeX) && (J < meshSizeY))
        {
            p_inMatrix[I][J].x = a_Comp.x;
            p_inMatrix[I][J].y = a_Comp.y;
        }
    }
}

void readIndxMapFromFile(std::map<pair < int, int >, int>& pMap, string p_FileName)
{
    string a_Fn = getCurrentDirectoryOnWindows() + "//" + p_FileName.c_str();
    ifstream inFile(a_Fn);
    CheckError(!inFile.fail(), "read file failed :" + p_FileName);
    while (!inFile.eof())
    {
        int I, J, a_Val;
        inFile >> I;
        inFile >> J;
        inFile >> a_Val;

        std::pair<int, int> a_KeyPair = std::make_pair(I,J);
        std::pair<pair < int, int >, int> a_MapPair = std::make_pair(a_KeyPair, a_Val);

        pMap.insert(a_MapPair);
    }
}


void read2DArrayFromFile(Complex **p_inMatrix, string p_FileName)
{
    string line, a_Temp;
    stringstream ss;
    ifstream inFile;
    inFile.open(getCurrentDirectoryOnWindows() + "//" + p_FileName.c_str());
    CheckError(!inFile.fail(), "read file failed");
    while (getline(inFile, line))
    {
        ss << line << "\r\n";
        Complex a_Comp;
        int I, J;
        ss >> a_Temp;
        a_Comp.x = std::stof(a_Temp);
        ss >> a_Temp;
        a_Comp.y = std::stof(a_Temp);
        ss >> a_Temp;
        I = std::stoi(a_Temp);
        ss >> a_Temp;
        J = std::stoi(a_Temp);
        p_inMatrix[I][J].x = a_Comp.x;
        p_inMatrix[I][J].y = a_Comp.y;
    }
}

void write2DArrayToFile(Complex(&p_inMatrix)[meshSizeX][meshSizeY], string p_FileName, int p_Scale)
{
    ofstream outFile;
    outFile.open(getCurrentDirectoryOnWindows() + "//" + p_FileName.c_str());
    CheckError(!outFile.fail(), "write file failed");
    outFile << setprecision(17);
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY; j++)
        {
            outFile << std::setw(17) << std::fixed << p_inMatrix[i][j].x;
            outFile << " ";
            outFile << std::setw(17) << std::fixed << p_inMatrix[i][j].y;
            outFile << " ";
            outFile << i;
            outFile << " ";
            outFile << j;
            outFile << "\n";
        }
    }
}


void write2DArrayToFile(Complex **p_inMatrix, int p_meshSizeX, int p_meshSizeY, string p_FileName)
{
    ofstream outFile;
    outFile.open(getCurrentDirectoryOnWindows() + "//" + p_FileName.c_str());
    CheckError(!outFile.fail(), "write file failed");
    for (int i = 0; i < p_meshSizeX; i++)
    {
        for (int j = 0; j < p_meshSizeY; j++)
        {
            outFile << p_inMatrix[i][j].x;
            outFile << " ";
            outFile << p_inMatrix[i][j].y;
            outFile << " ";
            outFile << i;
            outFile << " ";
            outFile << j;
            outFile << "\n";
        }
    }
}

bool CompareMatrix(Complex(&p_MatrixA)[meshSizeX][meshSizeY], Complex(&p_MatrixB)[meshSizeX][meshSizeY], string &pCompareError)
{
    char buffer[200];
    memset(buffer, 0, 200);
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY;j++)
        {
            if (p_MatrixA[i][j].x != p_MatrixB[i][j].x)
            {
                sprintf(buffer, "Xa %.*e <> Xb %.*e  I = %d  J = %d", FLT_DECIMAL_DIG - 1, p_MatrixA[i][j].x, FLT_DECIMAL_DIG - 1, p_MatrixB[i][j].x, i, j);
//                printf("Xa %.*e <> Xb %.*e\n", FLT_DECIMAL_DIG - 1, p_MatrixA[i][j].x, FLT_DECIMAL_DIG - 1, p_MatrixB[i][j].x);
//                printf("I = %d  J = %d\n", i, j);
                pCompareError = buffer;
                return false;
            }
            if (p_MatrixA[i][j].y != p_MatrixB[i][j].y)
            {
                sprintf(buffer, "Ya %.*e <> Yb %.*e  I = %d  J = %d", FLT_DECIMAL_DIG - 1, p_MatrixA[i][j].y, FLT_DECIMAL_DIG - 1, p_MatrixB[i][j].y, i, j);
 //               printf("Ya %.*e <> Yb %.*e\n", FLT_DECIMAL_DIG - 1, p_MatrixA[i][j].y, FLT_DECIMAL_DIG - 1, p_MatrixB[i][j].y);
 //               printf("I = %d  J = %d\n", i, j);
                pCompareError = buffer;

                return false;
            }

        }
    }
    return true;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t mulComplexWithCuda(float2* c, const float2* a, const float2* b, unsigned int size)
{
    Complex* dev_a = 0;
    Complex* dev_b = 0;
    Complex* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float2));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float2));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float2));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float2), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float2), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    mulKernelComplex << <1, size >> > (meshSize1D, dev_c, dev_a, dev_b, g_UseMultiplication);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mulKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float2), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}



int RunHeap1DTest()
{
    Initialization();
    size_t pitch;
    checkCudaErrors(hipMallocPitch((void**)&d_U, &pitch, meshSizeX * sizeof(Complex), meshSizeY * sizeof(Complex)));

    //    checkCudaErrors(hipMalloc((void**)&d_U, g_MatrixMemSize));
    ;

    Complex** a_h_dyn_array;
    a_h_dyn_array = new Complex * [meshSizeX];
    for (int i = 0; i < meshSizeX; i++)
    {
        a_h_dyn_array[i] = new Complex[meshSizeY];
        for (int j = 0; j < meshSizeY; j++)
        {
            a_h_dyn_array[i][j].x = 0;
            a_h_dyn_array[i][j].y = 0;
        }

    }

    printf("input array\n");
    read2DArrayFromFile(a_h_dyn_array, "input.txt");
    print2DArray(a_h_dyn_array, 4, 4);

    /*
        Complex* a_h_1Darray = new Complex[meshSizeX * meshSizeY];


        int a_Ind = 0;
        for (int i = 0; i < meshSizeX; i++)
        {
            for (int j = 0; j < meshSizeY; j++)
            {
                a_h_1Darray[a_Ind].x = a_h_dyn_array[i][j].x;
                a_h_1Darray[a_Ind].y = a_h_dyn_array[i][j].y;
                a_Ind++;
            }
        }
    */

    // Complex a_arr[meshSizeX][meshSizeY];
    int a_FWICount = 1;
    printf("input number of circles\n");
    cin >> a_FWICount;
    DWORD a_Start = GetTickCount();
    for (int i = 0; i < a_FWICount; i++)
    {

        //  checkCudaErrors(hipMemcpy2D(d_U, pitch, a_h_dyn_array, meshSizeX * sizeof(Complex), meshSizeX * sizeof(Complex), meshSizeY, hipMemcpyHostToDevice));

        checkCudaErrors(hipMemcpy(d_U, a_h_dyn_array, g_MatrixMemSize, hipMemcpyHostToDevice));

        //       checkCudaErrors(hipfftExecC2C(fftPlan, d_U, d_hMulResult, HIPFFT_FORWARD));
               //mulKernelComplex << <32, 256 >> > (d_hMulResult, d_U, d_Difract, g_UseMultiplication); //32
        //       checkCudaErrors(hipfftExecC2C(fftPlan, d_hMulResult, d_U, HIPFFT_BACKWARD));
               //checkCudaErrors(hipMemcpy(a_h_1Darray, d_U, g_MatrixMemSize, hipMemcpyDeviceToHost));
               //checkCudaErrors(hipMemcpy2D(a_h_dyn_array, meshSizeX * sizeof(Complex), d_hMulResult, pitch, meshSizeX * sizeof(Complex), meshSizeY, hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(&(a_h_dyn_array[0][0]), d_U, g_MatrixMemSize, hipMemcpyDeviceToHost));

    }
    printf("ExecC2CForwardInverse   Timing %d ms for %d circles\n", GetTickCount() - a_Start, a_FWICount);
    /*
    a_Ind = 0;
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY; j++)
        {
            a_h_dyn_array[i][j].x = a_h_1Darray[a_Ind].x;
            a_h_dyn_array[i][j].y = a_h_1Darray[a_Ind].y;
            a_Ind++;
        }
    }
    */
    printf("output array ExecC2CForward  Mul ExecC2Cinverse\n");
    print2DArray(a_h_dyn_array, 4, 4, 1);//meshSize1D


    for (int i = 0; i < meshSizeX; i++)
        delete[] a_h_dyn_array[i];
    delete[] a_h_dyn_array;




    return 0;
}

void CreateGenericMarix()
{
    int aSizeX = 8192;
    int aSizeY = 8192;
    Complex** a_h_dyn_array;
    a_h_dyn_array = new Complex * [aSizeX];
    for (int i = 0; i < aSizeX; i++)
    {
        a_h_dyn_array[i] = new Complex[aSizeY];
        for (int j = 0; j < aSizeY; j++)
        {
            a_h_dyn_array[i][j].x = 0;
            a_h_dyn_array[i][j].y = 0;
        }

    }

    write2DArrayToFile(a_h_dyn_array, aSizeX, aSizeY, "generic.txt");
    printf("generic.txt created:\n");
}


const Complex ComplexDiv(const Complex& x, const Complex& y)
{
    Complex temp;
    temp.x = ((x.x * y.x) + (x.y * y.y)) / (y.x * y.x + y.y * y.y);
    temp.y = ((x.y * y.x) - (x.x * y.y)) / (y.x * y.x + y.y * y.y);
    return temp;
}


void HostMatrixMultiplication(Complex(&p_Result)[meshSizeX][meshSizeY], Complex(&p_W)[meshSizeX][meshSizeY],
    Complex(&p_Difract)[meshSizeX][meshSizeY])
{
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY; j++)
        {
            p_Result[i][j].x = (p_W[i][j].x * p_Difract[i][j].x) - (p_W[i][j].y * p_Difract[i][j].y);
            p_Result[i][j].y = (p_W[i][j].x * p_Difract[i][j].y) + (p_W[i][j].y * p_Difract[i][j].x);

            //            printf("%.17g %.17g    [%d][%d] ==>>\n ",c[i][j].x, c[i][j].y, i, j);

            //            printf("W[%d][%d].x = %.17g  y = %.17g  *  D[%d][%d].x = %.17g  y = %.17g\n\n", 
            //              i, j, a[i][j].x, a[i][j].y, i, j, p_Difract[i][j].x, p_Difract[i][j].y);


        }
    }

}

void LogMatrixMultiplication(Complex(&p_Host)[meshSizeX][meshSizeY], Complex(&p_Device)[meshSizeX][meshSizeY])
{
    for (int i = 0; i < meshSizeX; i++)
    {
        for (int j = 0; j < meshSizeY; j++)
        {
            printf("HOST     %.17g  %.17g   [%d][%d] ==>>\n ", p_Host[i][j].x, p_Host[i][j].y, i, j);
            printf("DEVICE   %.17g  %.17g   [%d][%d] ==>>\n\n ", p_Device[i][j].x, p_Device[i][j].y, i, j);               
        }
    }
}

/*double g_test_X1 = 2.7311369767301008E-86;
double g_test_Y1 = 0.01663909886172361;

double g_test_X2 = 5.615324458278521E-47;
double g_test_Y2 = 0.000001;
        if (g_LogMultiplicationHost)
        {
            printf("Complex multiplication test\n");
            printf("X1 = %.17g  Y1 = %.17g\n", g_test_X1,  g_test_Y1);
            printf("X2 = %.17g  Y2 = %.17g\n", g_test_X2, g_test_Y2);

//            c[i].x = (a[i].x * b[i].x) - (a[i].y * b[i].y);
//            c[i].y = (a[i].x * b[i].y) + (a[i].y * b[i].x);

            double a_X3 = (g_test_X1 * g_test_X2) - (g_test_Y1 * g_test_Y2);
            double a_Y3 = (g_test_X1 * g_test_Y2) + (g_test_Y1 * g_test_X2);

            printf("Mul result: X3 = %.17g  Y3 = %.17g\n", a_X3, a_Y3);

        }


*/

